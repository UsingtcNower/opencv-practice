#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <opencv2/opencv.hpp>

#define SHARED_MEMORY_BANKS 16

cv::Mat histImg;

__global__ void hist64binKernel()
{}

__global__ void hist64binMerge()
{}

int main(int argc, char** argv)
{
    int histSize = 64;
    cv::Mat img0 = cv::imread("lena.png");
    if (img0.empty()) {
        printf("error imread.\n");
        return -1;
    }
    assert(img0.channels() == 3);
    cv::Mat img(img0.cols, img0.rows, CV_8UC1);
    cv::cvtColor(img0, img, CV_RGB2GRAY);

    // init
    int devId = findCudaDevice(argc, (const char**)argv);
    int* drData = NULL;
    checkCudaErrors(hipMalloc((void**)&drData, img.cols*img.rows*sizeof(int)));
    int* hist = NULL;
    checkCudaErrors(hipMalloc((void**)&hist, histSize*sizeof(int)));

    // calculate


    // draw
    int hist_w = 512;
    int hist_h = 400;
    histImg = cv::Mat(hist_w, hist_h, CV_8UC3, cv::Scalar(0,0,0));
    int bin_w = cvRound((double)hist_w/histSize);

    for (int i = 0; i < histSize; ++i)
    {
        hist[i] = hist[i] * hist_h / 256;
    }
    for (int i = 1; i < histSize; ++i) {
        cv::line(histImg, Point((i-1)*bin_w, hist_h-hist[i-1]),
                          Point(i*bin_w, hist_h-hist[i]),
                          cv::Scalar(255, 0, 0), 2, 8, 0);
    }
    cv::imwrite(histImg, "histImg.png");
    return 0;
}