#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>
#include <opencv2/opencv.hpp>

#define SHARED_MEMORY_BANKS 16
#define THREADBLOCK_SIZE SHARED_MEMORY_BANKS*4

cv::Mat histImg;

__global__ void vectorAddKernel(float* A, float* B, float* C, int numCnt)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < numCnt) {
        C[i] = A[i] + B[i];
    }
}

// partialHist[64][GridDim.x]
__global__ void hist64binKernel(uchar* drData, int* partialHist, int dataCount)
{
    __shared__ int s_hist[64*THREADBLOCK_SIZE];
    for (int i = 0; i < 64; ++i) {
        s_hist[i*THREADBLOCK_SIZE + threadIdx.x] = 0;
    }
    __syncthreads();
    int threadPos = blockDim.x*blockIdx.x + threadIdx.x;
    if (threadPos < dataCount) {
        s_hist[drData[threadPos] * THREADBLOCK_SIZE + threadIdx.x] ++;
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        for (int i = 0; i < 64; ++i) {
            int sum = 0;
            for (int j = 0; j < blockDim.x; j++) {
                sum += s_hist[i*THREADBLOCK_SIZE+j];
            }
            partialHist[i*gridDim.x + blockIdx.x] = sum;
        }
    }
}

__global__ void hist64binMerge(int* partialHist, int* hist, int histCount)
{
    __shared__ int data[256];
    int sum = 0;
    for (int i = threadIdx.x; i < histCount; i += 256) {
        sum += partialHist[blockIdx.x*histCount+i];
    }
    data[threadIdx.x] = sum;
    __syncthreads();
    if (threadIdx.x == 0) {
        sum = 0;
        for (int i = 0; i < 256; ++i)
            sum += data[i];
        hist[blockIdx.x] = sum;
    }
}

int main(int argc, char** argv)
{
    int histSize = 64;
    cv::Mat img0 = cv::imread("lena.png");
    if (img0.empty()) {
        printf("error imread.\n");
        return -1;
    }
    assert(img0.channels() == 3);
    cv::Mat img(img0.cols, img0.rows, CV_8UC1);
    cv::cvtColor(img0, img, CV_RGB2GRAY);

    // init
    int devId = findCudaDevice(argc, (const char**)argv);
    uchar* drData = NULL;
    checkCudaErrors(hipMalloc((void**)&drData, img.cols*img.rows*sizeof(uchar)));
    checkCudaErrors(hipMemcpy(drData, img.data, img.cols*img.rows*sizeof(uchar), hipMemcpyHostToDevice));
    int* d_hist = NULL;
    checkCudaErrors(hipMalloc((void**)&d_hist, histSize*sizeof(int)));
    dim3 dimBlock(THREADBLOCK_SIZE, 1);
    dim3 dimGrid((img.cols*img.rows+dimBlock.x-1)/dimBlock.x, 1);
    int* partialHist = NULL;
    checkCudaErrors(hipMalloc((void**)&partialHist, dimGrid.x*histSize*sizeof(int)));
    
    
    // calculate
    hist64binKernel <<<dimGrid, dimBlock >>>(drData, partialHist, img.cols*img.rows);
    hipDeviceSynchronize();
    getLastCudaError("hist64binKernel");
    // -->> paritalHist[64][dimGrid.x]
    hist64binMerge <<< histSize, 256>>>(partialHist, d_hist, dimGrid.x);
    hipDeviceSynchronize();
    getLastCudaError("hist64binMerge");

    int* hist = (int*)malloc(histSize*sizeof(int));
    checkCudaErrors(hipMemcpy(hist, d_hist, histSize*sizeof(int), hipMemcpyDeviceToHost));
    // draw
    int hist_w = 512;
    int hist_h = 400;
    histImg = cv::Mat(hist_w, hist_h, CV_8UC3, cv::Scalar(0,0,0));
    int bin_w = cvRound((double)hist_w/histSize);

    for (int i = 0; i < histSize; ++i)
    {
        hist[i] = hist[i] * hist_h / 256;
    }
    for (int i = 1; i < histSize; ++i) {
        cv::line(histImg, cv::Point((i-1)*bin_w, hist_h-hist[i-1]),
                          cv::Point(i*bin_w, hist_h-hist[i]),
                          cv::Scalar(255, 0, 0), 2, 8, 0);
    }
    cv::imwrite("histImg.png", histImg);
    return 0;
}