#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>
#include <opencv2/opencv.hpp>

#define SHARED_MEMORY_BANKS 16
#define THREADBLOCK_SIZE SHARED_MEMORY_BANKS*4

cv::Mat histImg;

__global__ void vectorAddKernel(float* A, float* B, float* C, int numCnt)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < numCnt) {
        C[i] = A[i] + B[i];
    }
}

__global__ void hist64binKernel(uchar* drData, int* partialHist)
{
    __shared__ uchar s_hist[64*THREADBLOCK_SIZE];
    for (int i = 0; i < 64; ++i) {
        s_hist[i*THREADBLOCK_SIZE + threadIdx.x] = 0;
    }
    __syncthreads();
    int threadPos = blockDim.x*blockIdx.x + threadIdx.x;
    s_hist[drData[threadPos] * THREADBLOCK_SIZE + threadIdx.x] ++;
    __syncthreads();
    if (threadIdx.x == 0) {
        for (int i = 0; i < 64; ++i) {
            int sum = 0;
            for (int j = 0; j < blockDim.x; j++) {
                sum += s_hist[i*THREADBLOCK_SIZE+threadIdx.x];
            }
            partialHist[i*gridDim.x + blockIdx.x] = sum;
        }
    }
}

__global__ void hist64binMerge(int* partialHist, int* hist)
{}

int main(int argc, char** argv)
{
    int histSize = 64;
    cv::Mat img0 = cv::imread("lena.png");
    if (img0.empty()) {
        printf("error imread.\n");
        return -1;
    }
    assert(img0.channels() == 3);
    cv::Mat img(img0.cols, img0.rows, CV_8UC1);
    cv::cvtColor(img0, img, CV_RGB2GRAY);

    // init
    int devId = findCudaDevice(argc, (const char**)argv);
    uchar* drData = NULL;
    checkCudaErrors(hipMalloc((void**)&drData, img.cols*img.rows*sizeof(uchar)));
    checkCudaErrors(hipMemcpy(drData, img.data, img.cols*img.rows*sizeof(uchar), hipMemcpyHostToDevice));
    int* hist = NULL;
    checkCudaErrors(hipMalloc((void**)&hist, histSize*sizeof(int)));
    dim3 dimBlock(THREADBLOCK_SIZE, 1);
    dim3 dimGrid((img.cols*img.rows+dimBlock.x-1)/dimBlock.x, 1);
    int* partialHist = NULL;
    checkCudaErrors(hipMalloc((void**)&partialHist, dimGrid.x*histSize*sizeof(int)));
    
    
    // calculate
    hist64binKernel << <dimGrid, dimBlock >> >(drData, partialHist);


    // draw
    int hist_w = 512;
    int hist_h = 400;
    histImg = cv::Mat(hist_w, hist_h, CV_8UC3, cv::Scalar(0,0,0));
    int bin_w = cvRound((double)hist_w/histSize);

    for (int i = 0; i < histSize; ++i)
    {
        hist[i] = hist[i] * hist_h / 256;
    }
    for (int i = 1; i < histSize; ++i) {
        cv::line(histImg, Point((i-1)*bin_w, hist_h-hist[i-1]),
                          Point(i*bin_w, hist_h-hist[i]),
                          cv::Scalar(255, 0, 0), 2, 8, 0);
    }
    cv::imwrite(histImg, "histImg.png");
    return 0;
}