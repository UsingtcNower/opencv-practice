#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <opencv2/opencv.hpp>

texture<uchar4, 2, hipReadModeElementType> rgbaTex;
uint* pImage;

__device__ uint uchar4Touint(uchar4 rgba)
{
    rgba.x = __saturatef(abs(rgba.x));
    rgba.y = __saturatef(abs(rgba.y));
    rgba.z = __saturatef(abs(rgba.z));
    rgba.w = __saturatef(abs(rgba.w));

    return uint(rgba.x) | (uint(rgba.y) << 8) | (uint(rgba.z) << 16) | (uint(rgba.w)<<24);
}

__global__ void dilate(uint *od, int w, int h, int r)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= w || y >= h) {
        return;
    }

    uchar4 center = tex2D(rgbaTex, x, y);
    uchar4 t = center;
    for (int i = -r; i <= r; ++i) {
        for (int j = -r; j <= r; ++j) {
            uchar4 curPix = tex2D(rgbaTex, x + i, y + i);
            if (t.x < curPix.x)
                t.x = curPix.x;
            if (t.y < curPix.y)
                t.y = curPix.y;
            if (t.z < curPix.z)
                t.z = curPix.z;
            if (t.w < curPix.w)
                t.w = curPix.w;
        }
    }
    //printf("%d,%d,%d,%d\n",t.x, t.y, t.z, t.w);
    od[y*w + x] = uchar4Touint(t);
    printf("%d\n", od[y*w+x]);
}

int main(int argc, char **argv)
{
    int width, height;
    int radius = 3;
    char *imagePath = "lena.png";
    cv::Mat image0 = cv::imread(imagePath);
    cv::Mat image(image0.cols, image0.rows, CV_8UC4);
    cv::cvtColor(image0, image, CV_RGB2RGBA);
    if (image.empty()) {
        printf("failed to read image.\n");
        exit(-1);
    }
    assert(image.channels() == 4);
    width = image.cols;
    height = image.rows;

    int devId = findCudaDevice(argc, (const char **)argv);

    // device memory for result 
    uint* dData = NULL;
    checkCudaErrors(hipMalloc((void **)&dData, width*height*sizeof(uint)));

    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc<uchar4>();
    hipArray *cuArray = NULL;
    checkCudaErrors(hipMallocArray(&cuArray, &channelDesc, width, height));
    checkCudaErrors(hipMemcpyToArray(cuArray, 0, 0, image.data, width*height*sizeof(uint), hipMemcpyHostToDevice));

    rgbaTex.addressMode[0] = hipAddressModeWrap;
    rgbaTex.addressMode[1] = hipAddressModeWrap;

    checkCudaErrors(hipBindTextureToArray(rgbaTex, cuArray, channelDesc));

    dim3 dimBlock(16,16);
    dim3 dimGrid((width+dimBlock.x-1)/dimBlock.x, (height+dimBlock.y-1)/dimBlock.y);
    dilate <<<dimGrid, dimBlock, 0>>>(dData, width, height, radius);

    checkCudaErrors(hipDeviceSynchronize());

    // host memory
    checkCudaErrors(hipMemcpy(image.data, dData, width*height*sizeof(uint), hipMemcpyDeviceToHost));

    // save
    cv::imwrite("me_dilate3.bmp", image);

    checkCudaErrors(hipFree(dData));
    checkCudaErrors(hipFreeArray(cuArray));
}