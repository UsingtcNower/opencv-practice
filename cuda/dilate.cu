#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <helper_functions.h>

texture<uchar4, 2, hipReadModeNormalizedFloat> rgbaTex;
uint* pImage;

__device__ uint float4Touint(float4 rgba)
{
    rgba.x = __saturatef(fabs(rgba.x));
    rgba.y = __saturatef(fabs(rgba.y));
    rgba.z = __saturatef(fabs(rgba.z));
    rgba.w = __saturatef(fabs(rgba.w));

    return uint(rgba.x*255.0f) | (uint(rgba.y*255.0f) << 8) | (uint(rgba.z*255.0f) << 16) | (uint(rgba.w*255.0f)<<24);
}

__global__ void dilate(uint *od, int w, int h, int r)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= w || y >= h) {
        return;
    }

    float4 center = tex2D(rgbaTex, x, y);
    float4 t = center;
    for (int i = -r; i <= r; ++i) {
        for (int j = -r; j <= r; ++j) {
            float4 curPix = tex2D(rgbaTex, x + i, y + i);
            if (t.x < curPix.x)
                t.x = curPix.x;
            if (t.y < curPix.y)
                t.y = curPix.y;
            if (t.z < curPix.z)
                t.z = curPix.z;
            if (t.w < curPix.w)
                t.w = curPix.w;
        }
    }
    od[y*w + x] = float4Touint(t);
}

int main(int argc, char **argv)
{
    int width, height;
    char *imagePath = "";
    LoadBMPFile();
}