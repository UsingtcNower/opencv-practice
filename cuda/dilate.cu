#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <helper_functions.h>

texture<uchar4, 2, hipReadModeNormalizedFloat> rgbaTex;
uint* pImage;

__device__ uint float4Touint(float4 rgba)
{
    rgba.x = __saturatef(fabs(rgba.x));
    rgba.y = __saturatef(fabs(rgba.y));
    rgba.z = __saturatef(fabs(rgba.z));
    rgba.w = __saturatef(fabs(rgba.w));

    return uint(rgba.x*255.0f) | (uint(rgba.y*255.0f) << 8) | (uint(rgba.z*255.0f) << 16) | (uint(rgba.w*255.0f)<<24);
}

__global__ void dilate(uint *od, int w, int h, int r)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= w || y >= h) {
        return;
    }

    float4 center = tex2D(rgbaTex, x, y);
    float4 t = center;
    for (int i = -r; i <= r; ++i) {
        for (int j = -r; j <= r; ++j) {
            float4 curPix = tex2D(rgbaTex, x + i, y + i);
            if (t.x < curPix.x)
                t.x = curPix.x;
            if (t.y < curPix.y)
                t.y = curPix.y;
            if (t.z < curPix.z)
                t.z = curPix.z;
            if (t.w < curPix.w)
                t.w = curPix.w;
        }
    }
    od[y*w + x] = float4Touint(t);
}

int main(int argc, char **argv)
{
    int width, height;
    char *imagePath = "";
    LoadBMPFile((uchar4 **)&pImage, &width, &height, imagePath);
    if (!pImage) {
        printf("failed top load bmp.\n");
        exit(-1);
    }

    int devId = findCudaDevice(argc, (const char **)argv);

    // result 
    uint* dData = NULL;
    checkCudaErrors(hipMalloc((void **)&dData, width*height*sizeof(uint)));

    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc<uchar4>();
    hipArray *cuArray = NULL;
    checkCudaErrors(hipMallocArray(&cuArray, channelDesc, width, height));
    checkCudaErrors(hipMemcpyToArray(cuArray, 0, 0, pImage, width*height*sizeof(uint), hipMemcpyHostToDevice));

    rgbaTex.addressMode[0] = hipAddressModeWrap;
    rgbaTex.addressMode[1] = hipAddressModeWrap;

    checkCudaErrors(hipBindTextureToArray(rgbaTex, cuArray, channelDesc);

}