#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <opencv2/opencv.hpp>

texture<uchar4, 2, hipReadModeNormalizedFloat> rgbaTex;
uint* pImage;

__device__ uint float4Touint(float4 rgba)
{
    rgba.x = __saturatef(fabs(rgba.x));
    rgba.y = __saturatef(fabs(rgba.y));
    rgba.z = __saturatef(fabs(rgba.z));
    rgba.w = __saturatef(fabs(rgba.w));

    return uint(rgba.x*255.0f) | (uint(rgba.y*255.0f) << 8) | (uint(rgba.z*255.0f) << 16) | (uint(rgba.w*255.0f)<<24);
}

__global__ void dilate(uint *od, int w, int h, int r)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= w || y >= h) {
        return;
    }

    float4 center = tex2D(rgbaTex, x, y);
    float4 t = center;
    for (int i = -r; i <= r; ++i) {
        for (int j = -r; j <= r; ++j) {
            float4 curPix = tex2D(rgbaTex, x + i, y + i);
            if (t.x < curPix.x)
                t.x = curPix.x;
            if (t.y < curPix.y)
                t.y = curPix.y;
            if (t.z < curPix.z)
                t.z = curPix.z;
            if (t.w < curPix.w)
                t.w = curPix.w;
        }
    }
    od[y*w + x] = float4Touint(t);
}

int main(int argc, char **argv)
{
    int width, height;
    int radius = 3;
    char *imagePath = "lena.png";
    cv::Mat image0 = cv::imread(imagePath);
    cv::Mat image(image0.cols, image0.rows, CV_8UC4);
    cv::cvtColor(image0, image, CV_RGB2RGBA);
    if (image.empty()) {
        printf("failed to read image.\n");
        exit(-1);
    }
    assert(image.channels() == 4);
    width = image.cols;
    height = image.rows;

    int devId = findCudaDevice(argc, (const char **)argv);

    // device memory for result 
    uint* dData = NULL;
    checkCudaErrors(hipMalloc((void **)&dData, width*height*sizeof(uint)));

    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray *cuArray = NULL;
    checkCudaErrors(hipMallocArray(&cuArray, &channelDesc, width, height));
    checkCudaErrors(hipMemcpyToArray(cuArray, 0, 0, image.data, width*height*sizeof(uint), hipMemcpyHostToDevice));

    rgbaTex.addressMode[0] = hipAddressModeWrap;
    rgbaTex.addressMode[1] = hipAddressModeWrap;

    checkCudaErrors(hipBindTextureToArray(rgbaTex, cuArray, channelDesc));

    dim3 dimBlock(16,16);
    dim3 dimGrid((width+dimBlock.x-1)/dimBlock.x, (height+dimBlock.y-1)/dimBlock.y);
    dilate <<<dimGrid, dimBlock, 0>>>(dData, width, height, radius);

    checkCudaErrors(hipDeviceSynchronize());

    // host memory
    checkCudaErrors(hipMemcpy(image.data, dData, width*height*sizeof(uint), hipMemcpyDeviceToHost));

    // save
    cv::imwrite("me_dilate3.bmp", image);

    checkCudaErrors(hipFree(dData));
    checkCudaErrors(hipFreeArray(cuArray));
}