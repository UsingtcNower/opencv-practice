#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

void sum_gold(int* a, int n, int* sum)
{
    *sum = 0;
    for (int i = 0; i < n; ++i)
        *sum += a[i];
}

#define THREAD_BLOCK_SIZE 256
#define MUL(a,b) ((a)*(b))
#define MUD(a,b,c) (MUL(a,b)+(c))

__global__ void getPartialSum_GPU(int* a, int n, int* partialSum)
{
    __shared__ int subSum[THREAD_BLOCK_SIZE];

    subSum[threadIdx.x] = 0;
    __syncthreads();

    for (int i = MUD(blockIdx.x, blockDim.x, threadIdx.x); i < n; i += MUL(gridDim.x,blockDim.x)) {
        subSum[threadIdx.x] += a[i];
    }
    __syncthreads();

#if 0
    for (int stride = THREAD_BLOCK_SIZE / 2; stride > 0; stride = stride >> 1) {
        if (threadIdx.x < stride) {
            subSum[threadIdx.x] += subSum[threadIdx.x + stride];
            __syncthreads();
        }
    }
    partialSum[blockIdx.x] = subSum[0];
#else
    if (threadIdx.x == 0) {
        for (int i = 0; i < THREAD_BLOCK_SIZE;i++) {
            partialSum[blockIdx.x] += subSum[i];
        }
    }
#endif

}

#define ARRAY_SIZE (1<<20LL)

int main()
{
    int *h_a = (int*)malloc(ARRAY_SIZE*sizeof(int));
    int n = ARRAY_SIZE;
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; ++i) {
        h_a[i] = rand() % 2;
    }
    int h_sum = 0;
    sum_gold(h_a, ARRAY_SIZE, &h_sum);

    int* d_a;
    checkCudaErrors(hipMalloc((void**)&d_a, ARRAY_SIZE*sizeof(int)));
    checkCudaErrors(hipMemcpy(d_a, h_a, ARRAY_SIZE*sizeof(int), hipMemcpyHostToDevice));
    
    dim3 dimBlock(THREAD_BLOCK_SIZE);
    dim3 dimGrid(ARRAY_SIZE/THREAD_BLOCK_SIZE);
    int* dPartial_sum;
    checkCudaErrors(hipMalloc((void**)&dPartial_sum, dimGrid.x*sizeof(int)));
    getPartialSum_GPU << <dimGrid, dimBlock >> >(d_a, ARRAY_SIZE, dPartial_sum);

    int* hPartial_sum = (int *)malloc(sizeof(int)*dimGrid.x);
    checkCudaErrors(hipMemcpy(hPartial_sum, dPartial_sum, dimGrid.x*sizeof(int), hipMemcpyDeviceToHost));

    int d_sum = 0;
    for (int i = 0; i < dimGrid.x; ++i) {
        d_sum += hPartial_sum[i];
    }

    printf("Check Result: %d\n", d_sum == h_sum);
    //getchar();
}