#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

int main(int argv, char** argc)
{
    int devId;
    hipDeviceProp_t devProp;
    checkCudaErrors(hipGetDevice(&devId));
    checkCudaErrors(hipGetDeviceProperties(&devProp, devId));
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devId, devProp.name, devProp.major, devProp.minor);
}