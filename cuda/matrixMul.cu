#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <time.h>

typedef struct{
	int width,height;
	int pitch;
	int* pdata;
} Matrix;

void matrixMul_Gold(Matrix A, Matrix B, Matrix C)
{
	for(int y=0;y<C.height;y++) {
		for(int x=0;x<C.width;++x) {
			int sum = 0;
			for(int i=0;i<A.width;++i)
				sum += A.pdata[y*A.width+i]*B.pdata[i*B.width+x];
			C.pdata[y*C.width+x] = sum;
		}
	}
}

__global__ void matrixMul(Matrix d_A, Matrix d_B, Matrix d_C)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	if(x<d_C.width && y<d_C.height) {
		int sum = 0;
		for(int i=0;i<d_A.width;++i)
			sum += d_A.pdata[y*d_A.width+i]*d_B.pdata[i*d_B.width+x];
		d_C.pdata[y*d_C.width+x] = sum;
	}	
}

__device__ Matrix getSubMatrix(Matrix A, int x, int y)
{
    Matrix a;
    a.width = 16;
    a.height = 16;
    a.pitch = a.pitch;
    a.pdata = &A.pdata[y*16*A.width+x*16];
    return a;
}

__global__ void matrixMul_SharedMemory(Matrix d_A, Matrix d_B, Matrix d_C)
{
    int blockX = blockIdx.x;
    int blockY = blockIdx.y;

    Matrix cSub = getSubMatrix(d_C, blockX, blockY);
    int x = threadIdx.x;
    int y = threadIdx.y;

    int cSum = 0;

    for (int i = 0; i < (d_A.width / 16); ++i) {
        __shared__ int sA[16][16];
        __shared__ int sB[16][16];

        Matrix aSub = getSubMatrix(d_A, i, blockY);
        Matrix bSub = getSubMatrix(d_B, blockX, i);

        sA[y][x] = aSub.pdata[y*d_A.width+x];
        sB[y][x] = bSub.pdata[y*d_B.width+x];

        __syncthreads();

        for (int j = 0; j < 16; ++j)
            cSum += sA[y][j] * sB[j][x];

        __syncthreads();
    }

    cSub.pdata[y*d_C.width+x] = cSum;
}

int main(int argc, char** argv)
{
	Matrix A;
    A.width = 16;// 720;
    A.height = 16;// 640;
    A.pitch = A.width*sizeof(int);
	A.pdata = (int*)malloc(A.width*A.height*sizeof(int));

	Matrix B;
    B.width = 32;//1280;
    B.height = 16;// 720;
    B.pitch = B.width*sizeof(int);
	B.pdata = (int*)malloc(B.width*B.height*sizeof(int));

	Matrix C;
	C.width = B.width;
	C.height = A.height;
    C.pitch = C.width*sizeof(int);
	C.pdata = (int*)malloc(C.width*C.height*sizeof(int));

	srand((unsigned)time(NULL));
	for(int y=0;y<A.height;++y)
		for(int x=0;x<A.width;++x)
			A.pdata[y*A.width+x] = rand()%256;
	for(int y=0;y<B.height;++y)
		for(int x=0;x<B.width;++x)
			B.pdata[y*B.width+x] = rand()%256;
	matrixMul_Gold(A,B,C);

	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
    d_A.pitch = A.pitch;
	checkCudaErrors(hipMalloc((void**)&d_A.pdata, d_A.width*d_A.height*sizeof(int)));
	checkCudaErrors(hipMemcpy(d_A.pdata, A.pdata, A.width*A.height*sizeof(int), hipMemcpyHostToDevice));

	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
    d_B.pitch = B.pitch;
	checkCudaErrors(hipMalloc((void**)&d_B.pdata, d_B.width*d_B.height*sizeof(int)));
	checkCudaErrors(hipMemcpy(d_B.pdata, B.pdata, B.width*B.height*sizeof(int), hipMemcpyHostToDevice));

	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
    d_C.pitch = C.pitch;
	checkCudaErrors(hipMalloc((void**)&d_C.pdata, d_C.width*d_C.height*sizeof(int)));

	Matrix h_C;
	h_C.width = C.width;
	h_C.height = C.height;
    h_C.pitch = C.pitch;
	h_C.pdata = (int*)malloc(h_C.width*h_C.height*sizeof(int));

	dim3 dimBlock(16, 16);
	dim3 dimGrid((d_C.width+dimBlock.x-1)/dimBlock.x, (d_C.height+dimBlock.y-1)/dimBlock.y);
	//matrixMul<<<dimGrid, dimBlock >>>(d_A, d_B, d_C);
    matrixMul_SharedMemory<<<dimGrid, dimBlock >>>(d_A, d_B, d_C);
	
	checkCudaErrors(hipMemcpy(h_C.pdata, d_C.pdata, d_C.width*d_C.height*sizeof(int), hipMemcpyDeviceToHost));

	bool check = true;
	for(int y=0;y<h_C.height && check;++y)
		for(int x=0;x<h_C.width && check;++x)
			if(h_C.pdata[y*h_C.width+x]!=C.pdata[y*h_C.width+x]) {
				check = false;
                printf("%d, %d, %d, %d\n", y, x, h_C.pdata[y*h_C.width + x], C.pdata[y*h_C.width + x]);
				break;
			}

    hipFree(d_A.pdata);
    hipFree(d_B.pdata);
    hipFree(d_C.pdata);

	printf("Check Result: %d\n", check);
	return 0;
}
