//#include <cuda_runtime.h>
//#include <helper_cuda.h>

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

typedef struct{
	int width,height;
	int* pdata;
} Matrix;

void matrixMul_Gold(Matrix A, Matrix B, Matrix C)
{
	for(int y=0;y<C.height;y++) {
		for(int x=0;x<C.width;++x) {
			int sum = 0;
			for(int i=0;i<A.width;++i)
				sum += A.pdata[y*A.width+i]*B.pdata[i*B.width+x];
			C.pdata[y*C.width+x] = sum;
		}
	}
}

int main(int argc, char** argv)
{
	Matrix A;
	A.width = 360;
	A.height = 640;
	A.pdata = (int*)malloc(A.width*A.height*sizeof(int));

	Matrix B;
	B.width = 1280;
	B.height = 360;
	B.pdata = (int*)malloc(B.width*B.height*sizeof(int));

	Matrix C;
	C.width = B.width;
	C.height = A.height;
	C.pdata = (int*)malloc(C.width*C.height*sizeof(int));

	srand((unsigned)time(NULL));
	for(int y=0;y<A.height;++y)
		for(int x=0;x<A.width;++x)
			A.pdata[y*A.width+x] = rand()%256;
	for(int y=0;y<B.height;++y)
		for(int x=0;x<B.width;++x)
			B.pdata[y*B.width+x] = rand()%256;
	matrixMul_Gold(A,B,C);
	return 0;
}
