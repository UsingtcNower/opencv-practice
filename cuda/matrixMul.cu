#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <time.h>

typedef struct{
	int width,height;
	int pitch;
	int* pdata;
} Matrix;

void matrixMul_Gold(Matrix A, Matrix B, Matrix C)
{
	for(int y=0;y<C.height;y++) {
		for(int x=0;x<C.width;++x) {
			int sum = 0;
			for(int i=0;i<A.width;++i)
				sum += A.pdata[y*A.width+i]*B.pdata[i*B.width+x];
			C.pdata[y*C.width+x] = sum;
		}
	}
}

__global__ void matrixMul(Matrix d_A, Matrix d_B, Matrix d_C)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	if(x<d_C.width && y<d_C.height) {
		int sum = 0;
		for(int i=0;i<d_A.width;++i)
			sum += d_A.pdata[y*d_A.width+i]*d_B.pdata[i*d_B.width+x];
		d_C.pdata[y*d_C.width+x] = sum;
	}	
}

int main(int argc, char** argv)
{
	Matrix A;
	A.width = 360;
	A.height = 640;
	A.pdata = (int*)malloc(A.width*A.height*sizeof(int));

	Matrix B;
	B.width = 1280;
	B.height = 360;
	B.pdata = (int*)malloc(B.width*B.height*sizeof(int));

	Matrix C;
	C.width = B.width;
	C.height = A.height;
	C.pdata = (int*)malloc(C.width*C.height*sizeof(int));

	srand((unsigned)time(NULL));
	for(int y=0;y<A.height;++y)
		for(int x=0;x<A.width;++x)
			A.pdata[y*A.width+x] = rand()%256;
	for(int y=0;y<B.height;++y)
		for(int x=0;x<B.width;++x)
			B.pdata[y*B.width+x] = rand()%256;
	matrixMul_Gold(A,B,C);

	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	checkCudaErrors(hipMalloc((void**)&d_A.pdata, d_A.width*d_A.height*sizeof(int)));
	checkCudaErrors(hipMemcpy(d_A.pdata, A.pdata, A.width*A.height*sizeof(int), hipMemcpyHostToDevice));

	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	checkCudaErrors(hipMalloc((void**)&d_B.pdata, d_B.width*d_B.height*sizeof(int)));
	checkCudaErrors(hipMemcpy(d_B.pdata, B.pdata, B.width*B.height*sizeof(int), hipMemcpyHostToDevice));

	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	checkCudaErrors(hipMalloc((void**)&d_C.pdata, d_C.width*d_C.height*sizeof(int)));

	Matrix h_C;
	h_C.width = C.width;
	h_C.height = C.height;
	h_C.pdata = (int*)malloc(h_C.width*h_C.height*sizeof(int));

	dim3 dimBlock(16, 16);
	dim3 dimGrid((d_C.width+dimBlock.x-1)/dimBlock.x, (d_C.height+dimBlock.y-1)/dimBlock.y);
	matrixMul<<<dimGrid, dimBlock >>>(d_A, d_B, d_C);
	
	checkCudaErrors(hipMemcpy(h_C.pdata, d_C.pdata, d_C.width*d_C.height*sizeof(int), hipMemcpyDeviceToHost));

	bool check = true;
	for(int y=0;y<h_C.height;++y)
		for(int x=0;x<h_C.width;++x)
			if(h_C.pdata[y*h_C.width+x]!=C.pdata[y*h_C.width+x]) {
				check = false;
				break;
			}

    hipFree(d_A.pdata);
    hipFree(d_B.pdata);
    hipFree(d_C.pdata);

	printf("Check Result: %d\n", check);
	return 0;
}
